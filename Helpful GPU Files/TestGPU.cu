/* To compile: nvcc TestGPU.cu -o temp -lcudart -run
*/

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <math.h>

#define N (unsigned int)(1000000000)
unsigned char *A_CPU, *B_CPU, *C_CPU; 

unsigned char *A_GPU, *B_GPU, *C_GPU; 

dim3 dimBlock; 
dim3 dimGrid;

void AllocateMemory()
{	
	hipMalloc(&A_GPU,N*sizeof(unsigned char));
	hipMalloc(&B_GPU,N*sizeof(unsigned char));
	hipMalloc(&C_GPU,N*sizeof(unsigned char));

	A_CPU = (unsigned char*)malloc(N*sizeof(unsigned char));
	B_CPU = (unsigned char*)malloc(N*sizeof(unsigned char));
	C_CPU = (unsigned char*)malloc(N*sizeof(unsigned char));
}

void Innitialize()
{
	dimBlock.x = 1024;
	int blocks = (N+1023)/1024;
	dimGrid.x = (int)pow(blocks,1.0/3.0) + 1;
	dimGrid.y = (int)pow(blocks,1.0/3.0) + 1;
	dimGrid.z = (int)pow(blocks,1.0/3.0) + 1; 
	int i;
	
	for(i = 0; i < N; i++)
	{		
		A_CPU[i] = (unsigned char)1;	
		B_CPU[i] = (unsigned char)1;
		C_CPU[i] = (unsigned char)0;
	}
}

unsigned long int Additup(unsigned char *C_CPU)
{
	unsigned long int temp = 0;
	for(int i =0; i<N; i++)
	{
		temp += C_CPU[i];
	}
	return(temp);
}

void CleanUp(unsigned char *A_CPU,unsigned char*B_CPU,unsigned char *C_CPU,unsigned char*A_GPU,unsigned char *B_GPU,unsigned char *C_GPU)  //free
{
	free(A_CPU); free(B_CPU); free(C_CPU);
	hipFree(A_GPU); hipFree(B_GPU); hipFree(C_GPU);
}

__global__ void Addition(unsigned char *A, unsigned char *B, unsigned char *C)
{

	unsigned int id = threadIdx.x + blockDim.x*blockIdx.x + blockDim.x*gridDim.x*blockIdx.y +blockDim.x*gridDim.x*blockDim.y*gridDim.y*blockIdx.z;
	if(id < N)
	{
		C[id] = A[id]*B[id];
	}
}

int main()
{
	unsigned long int total;
	int i;
	timeval start, end;
	hipError_t err;
	
	AllocateMemory();

	Innitialize();
	
	gettimeofday(&start, NULL);

	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(unsigned char), hipMemcpyHostToDevice);
		
	Addition<<<dimGrid,dimBlock>>>(A_GPU, B_GPU, C_GPU);
		
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(unsigned char), hipMemcpyDeviceToHost);
	total = Additup(C_CPU);

	gettimeofday(&end, NULL);

	float time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	
	printf("Time in milliseconds= %.15f\n", (time/1000.0));	
	
	for(i = 0; i < N; i++)		
	{		
		//clearprintf("C[%d] = %d", i, C_CPU[i]);
	}
	int blocks = (N+1023)/1024;

	printf("Here she is %d %li %d\n",(int)pow(blocks,1.0/3.0) + 1 ,total, N);
	
	CleanUp(A_CPU,B_CPU,C_CPU,A_GPU,B_GPU,C_GPU);	
	
	return(0);
}
